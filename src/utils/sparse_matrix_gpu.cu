#include "hip/hip_runtime.h"
#include "sparse_matrix_gpu.h"
#include <thrust/device_vector.h>
#include <hipsparse.h>
#include <hipsolver.h>

using namespace Eigen;
using namespace mfem;

// CUDA API error checking
#define CUDA_CHECK(err)                                                                            \
    do {                                                                                           \
        hipError_t err_ = (err);                                                                  \
        if (err_ != hipSuccess) {                                                                 \
            printf("CUDA error %d at %s:%d\n", err_, __FILE__, __LINE__);                          \
            throw std::runtime_error("CUDA error");                                                \
        }                                                                                          \
    } while (0)

// cusolver API error checking
#define CUSOLVER_CHECK(err)                                                                        \
    do {                                                                                           \
        hipsolverStatus_t err_ = (err);                                                             \
        if (err_ != HIPSOLVER_STATUS_SUCCESS) {                                                     \
            printf("cusolver error %d at %s:%d\n", err_, __FILE__, __LINE__);                      \
            throw std::runtime_error("cusolver error");                                            \
        }                                                                                          \
    } while (0)

namespace {

  template<int N>
  struct psd_fix_functor {
      psd_fix_functor(double* V, double* W, double* A,
      double* Ainv) : V_(V), W_(W), A_(A), Ainv_(Ainv) {}

      __device__
      void operator()(int i) const {
          // Map<Matrix<double, N, N>> Ainvi(Ainv_ + i*N*N);
          Map<Matrix<double, N, N>> Vi(V_ + i*N*N);
          Map<Matrix<double, N, N>> Ai(A_ + i*N*N);
          Map<Matrix<double, N, 1>> Wi(W_ + i*N);
          
          Matrix<double, N, 1> Wi_inv;
          for (int j = 0; j < N; ++j) {
            // PSD fix
            if (Wi(j) < 1e-8) {
              Wi(j) = 1e-8;
            }
            Wi_inv(j) = 1.0 / Wi(j);
          }
          Ai = Vi * Wi.asDiagonal() * Vi.transpose();
          // Ainvi = Vi * Wi_inv.asDiagonal() * Vi.transpose();
          // printf("Wi %f", Wi(0));
          // printf("Ai %f", Ai(0, 0));
          // printf("Vi %f", Vi(0, 0));
      }

  private:
      double *V_;
      double *W_;
      double *A_;
      double *Ainv_;
  };

}

SparseMatrixGpu::SparseMatrixGpu(const SparseMatrix<double, RowMajor>& A) {
  init(A);
}

void SparseMatrixGpu::init(SparseMatrix<double, RowMajor> A) {
  A.makeCompressed();

  rows_ = A.rows();
  cols_ = A.cols();

  // Sparse matrix CSR data
  int nnz = A.nonZeros();
  hipMalloc((void**)&d_csr_row_offsets, sizeof(int) * (rows_ + 1));
  hipMalloc((void**)&d_csr_columns, sizeof(int) * nnz);
  hipMalloc((void**)&d_csr_values, sizeof(double) * nnz);

  // Copy CSR data to device
  hipMemcpy(d_csr_row_offsets, A.outerIndexPtr(),
      sizeof(int) * (rows_+1),
      hipMemcpyHostToDevice);
  hipMemcpy(d_csr_columns, A.innerIndexPtr(), sizeof(int) * nnz,
      hipMemcpyHostToDevice);
  hipMemcpy(d_csr_values, A.valuePtr(), sizeof(double) * nnz,
      hipMemcpyHostToDevice);

  // Create a cusparse handle and matrix descriptor
  hipsparseCreate(&handle);
  hipsparseCreateCsr(&matA, rows_, cols_, nnz,
      d_csr_row_offsets, d_csr_columns, d_csr_values,
      HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
      HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

  // Create dense output vector
  hipMalloc((void**)&d_y, sizeof(double) * rows_);

  // Create dense vector X & Y vectors
  hipsparseCreateDnVec(&vecY, rows_, d_y, HIP_R_64F);
}

void SparseMatrixGpu::product(double* dx, double** y) {
  hipsparseDnVecDescr_t vecX;
  hipsparseCreateDnVec(&vecX, cols_, dx, HIP_R_64F);

  // allocate an external buffer if needed
  if (dBuffer == NULL) {
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                            HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
  }

  // execute SpMV
  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
              &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
              HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);

  hipsparseDestroyDnVec(vecX);
  *y = d_y;
}

void SparseMatrixGpu::product(const double* dx, double** y) {
  hipsparseConstDnVecDescr_t vecX;
  hipsparseCreateConstDnVec(&vecX, cols_, dx, HIP_R_64F);

  // allocate an external buffer if needed
  if (dBuffer == NULL) {
    hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                            &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                            HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
    hipMalloc(&dBuffer, bufferSize);
  }

  // execute SpMV
  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
              &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
              HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);

  hipsparseDestroyDnVec(vecX);
  *y = d_y;
}

template<int N>
MatrixBatchInverseGpu<N>::MatrixBatchInverseGpu(int batch_size, double solver_tol,
    int max_sweeps) 
    : batch_size_(batch_size), solver_tol_(solver_tol), max_sweeps_(max_sweeps) {

  std::cout << "MatrixBatchInverseGpu::MatrixBatchInverseGpu()" << std::endl;
  std::vector<int> info(batch_size, 0); 

  /* step 1: create cusolver handle, bind a stream */
  CUSOLVER_CHECK(hipsolverDnCreate(&cusolverH));

  CUDA_CHECK(hipStreamCreateWithFlags(&stream, hipStreamNonBlocking));
  CUSOLVER_CHECK(hipsolverSetStream(cusolverH, stream));

  /* step 2: configuration of syevj */
  CUSOLVER_CHECK(hipsolverDnCreateSyevjInfo(&syevj_params));

  /* default value of tolerance is machine zero */
  CUSOLVER_CHECK(hipsolverDnXsyevjSetTolerance(syevj_params, solver_tol_));

  /* default value of max. sweeps is 100 */
  CUSOLVER_CHECK(hipsolverDnXsyevjSetMaxSweeps(syevj_params, max_sweeps_));

  /* disable sorting */
  CUSOLVER_CHECK(hipsolverDnXsyevjSetSortEig(syevj_params, sort_eig));

  /* step 3: create device vectors */
  int size_A = N * N * batch_size;
  int size_W = N * batch_size;
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_A), sizeof(double) * size_A));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_V), sizeof(double) * size_A));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_W), sizeof(double) * size_W));
  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_info), sizeof(int) * batch_size));

  /* step 4: query working space of syevj */
  CUSOLVER_CHECK(hipsolverDnDsyevjBatched_bufferSize(cusolverH, jobz, uplo, N,
      d_A, N, d_W, &lwork, syevj_params, batch_size));

  CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_work), sizeof(double) * lwork)); 
  std::cout << "MatrixBatchInverseGpu() done" << std::endl;
}

template<int N>
void MatrixBatchInverseGpu<N>::compute(double* A, double* Ainv) {
  // Copy device input to d_A (which will be overwritten)
  int size_A = N * N * batch_size_;
  int size_W = N * batch_size_;
  CUDA_CHECK(hipMemcpyAsync(d_A, A, sizeof(double) * size_A,
      hipMemcpyDeviceToDevice, stream));
  
  // Compute eigen-pairs. Eigenvectors are in d_A and eigenvalues in d_W
  CUSOLVER_CHECK(hipsolverDnDsyevjBatched(
      cusolverH, jobz, uplo, N, d_A, N, d_W, d_work, lwork,
      d_info, syevj_params, batch_size_));

  // std::vector<int> info(batch_size_, 0);
  // CUDA_CHECK(hipMemcpy(info.data(), d_info, sizeof(int) * batch_size_,
  //       hipMemcpyDeviceToHost));

  // std::vector<double> W(size_W, 0.0);
  // CUDA_CHECK(
  //       hipMemcpyAsync(W.data(), d_W, sizeof(double) * W.size(), hipMemcpyDeviceToHost, stream));
  // // OKAY we have eigenvector and eigenvalues. Now we compute
  // // the inverse
  CUDA_CHECK(hipStreamSynchronize(stream));

  // for (int i = 0; i < batch_size_; i++) {
  //   if (0 == info[i]) {
  //       // std::printf("matrix %d: syevj converges \n", i);
  //   } else if (0 > info[i]) {
  //       /* only info[0] shows if some input parameter is wrong.
  //         * If so, the error is HIPSOLVER_STATUS_INVALID_VALUE.
  //         */
  //       std::printf("Error: %d-th parameter is wrong \n", -info[i]);
  //       exit(1);
  //   } else { /* info = m+1 */
  //             /* if info[i] is not zero, Jacobi method does not converge at i-th matrix. */
  //       std::printf("WARNING: matrix %d, info = %d : sygvj does not converge \n", i, info[i]);
  //   }
  // }

  // for (int i = 0; i < W.size(); ++i) {
  //   std::cout << "W[i] " << W[i] << std::endl;
  // }

  thrust::for_each(thrust::counting_iterator<int>(0),
      thrust::counting_iterator<int>(batch_size_),
      psd_fix_functor<N>(d_A, d_W, A, Ainv));
}

template class mfem::MatrixBatchInverseGpu<6>; // 6x6 matrices
template class mfem::MatrixBatchInverseGpu<3>; // 6x6 matrices
