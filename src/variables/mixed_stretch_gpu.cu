#include "hip/hip_runtime.h"
#include "mixed_stretch_gpu.h"
#include "mesh/mesh.h"
#include <hipsparse.h>

using namespace Eigen;
using namespace mfem;
using namespace thrust::placeholders;

template<int DIM>
MixedStretchGpu<DIM>::MixedStretchGpu(std::shared_ptr<Mesh> mesh)
    : MixedVariable<DIM>(mesh) {
  nelem_ = mesh_->T_.rows();

  s_.resize(N()*nelem_);
  g_.resize(N()*nelem_);
  ds_.resize(N()*nelem_);
  la_.resize(N()*nelem_);
  // la_.setZero();
  R_.resize(M()*nelem_);
  S_.resize(N()*N()*nelem_);
  H_.resize(N()*N()*nelem_);
  dSdF_.resize(N()*M()*nelem_);
  Hinv_.resize(N()*N()*nelem_);
  Hloc_.resize(N()*N()*nelem_);

  std::cout << "Assuming triangles in 2D and tetrahedra in 3D" << std::endl;
  int N_loc; // size of local stiffness matrix
  if constexpr (DIM == 2) {
    N_loc = DIM * 3;
  } else {
    N_loc = DIM * 4;
  }
  Aloc_.resize(N_loc*N_loc*nelem_);
}

template<int DIM>
__device__ 
void MixedStretchGpu<DIM>::init_variables(int i, double* si_data) {
  Map<VecN> si(si_data + N()*i);
  if constexpr (DIM == 2) {
    si << 1, 1, 0;
  } else {
    si << 1, 1, 1, 0, 0, 0;
  }
}

template<int DIM>
void MixedStretchGpu<DIM>::reset() {

  double* si_data = thrust::raw_pointer_cast(s_.data());
  thrust::for_each(thrust::counting_iterator<int>(0),
      thrust::counting_iterator<int>(nelem_),
      [this, si_data] __device__ (const int i) {
        Map<VecN> si(si_data + N()*i);
        if constexpr (DIM == 2) {
            si << 1, 1, 0;
        } else {
            si << 1, 1, 1, 0, 0, 0;
        }
  });

  MatrixXd tmp = mesh_->V_.transpose();
  VectorXd x = Map<VectorXd>(tmp.data(), mesh_->V_.size());
  VectorXd def_grad;
  mesh_->deformation_gradient(x, def_grad);

  Eigen::SparseMatrix<double, RowMajor> A =
      mesh_->template jacobian<JacobianType::FULL>();
  A.makeCompressed();

  // Create a cusparse handle and matrix descriptor
  hipsparseHandle_t     handle = NULL;
  hipsparseSpMatDescr_t matA;
  hipsparseDnVecDescr_t vecX, vecY;
  void*                dBuffer    = NULL;
  size_t               bufferSize = 0;
  double alpha = 1.0f;
  double beta  = 0.0f;

  // Sparse matrix CSR data
  int nnz = A.nonZeros();
  int *d_csr_row_offsets, *d_csr_columns;
  double *d_csr_values;
  hipMalloc((void**)&d_csr_row_offsets, sizeof(int) * (A.rows() + 1));
  hipMalloc((void**)&d_csr_columns, sizeof(int) * nnz);
  hipMalloc((void**)&d_csr_values, sizeof(double) * nnz);

  // Copy CSR data to device
  hipMemcpy(d_csr_row_offsets, A.outerIndexPtr(), sizeof(int) * (A.rows()+1),
      hipMemcpyHostToDevice);
  hipMemcpy(d_csr_columns, A.innerIndexPtr(), sizeof(int) * nnz,
      hipMemcpyHostToDevice);
  hipMemcpy(d_csr_values, A.valuePtr(), sizeof(double) * nnz,
      hipMemcpyHostToDevice);

  // Create a cusparse handle and matrix descriptor
  hipsparseCreate(&handle);
  hipsparseCreateCsr(&matA, A.rows(), A.cols(), nnz,
                    d_csr_row_offsets, d_csr_columns, d_csr_values,
                    HIPSPARSE_INDEX_32I, HIPSPARSE_INDEX_32I,
                    HIPSPARSE_INDEX_BASE_ZERO, HIP_R_64F);

  // Create dense input vector
  double* dx;
  hipMalloc((void**)&dx, sizeof(double) * A.cols());
  hipMemcpy(dx, x.data(), sizeof(double) * A.cols(),
      hipMemcpyHostToDevice);
  // Create dense output vector
  double *dy;
  hipMalloc((void**)&dy, sizeof(double) * A.rows());

  // Create dense vector X & Y vectors
  hipsparseCreateDnVec(&vecX, A.cols(), dx, HIP_R_64F);
  hipsparseCreateDnVec(&vecY, A.rows(), dy, HIP_R_64F);

  // allocate an external buffer if needed
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
                          &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
                          HIPSPARSE_SPMV_ALG_DEFAULT, &bufferSize);
  hipMalloc(&dBuffer, bufferSize);

  // hipsparseStatus_t status;
  // FINALLY perform SPMV
  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE,
              &alpha, matA, vecX, &beta, vecY, HIP_R_64F,
              HIPSPARSE_SPMV_ALG_DEFAULT, dBuffer);

  // destroy matrix/vector descriptors
  hipsparseDestroySpMat(matA);
  hipsparseDestroyDnVec(vecX);
  hipsparseDestroyDnVec(vecY);
  hipsparseDestroy(handle);

  // Copy the result back to the host
  VectorXd F(A.rows());
  hipMemcpy(F.data(), dy, sizeof(double)*A.rows(), hipMemcpyDeviceToHost);
  VectorXd F2 = A * x;
  std::cout << "F: " << F << std::endl;
  // std::cout << "def_grad: " << def_grad << std::endl;
  std::cout << "F - def_grad: " << (F - def_grad).norm() << std::endl;
  std::cout << "F2 - def_grad: " << (F2 - def_grad).norm() << std::endl;
    std::cout << "x size: " << x.size() << " A rows: " << A.rows() << " A cols: " << A.cols() << std::endl;
    std::cout << "bufferSize" << bufferSize << std::endl;
  // Copy Eigen matrix to GPU with CUDA and perform sparse
  // matrix-vector multiplication with CUSPARSE

  hipFree(dBuffer);
  hipFree(d_csr_row_offsets);
  hipFree(d_csr_columns);
  hipFree(d_csr_values);
  hipFree(dx);
  hipFree(dy);


  // std::cout << " Nelem: " << nelem_ << 
  // Aloc_.resize(nelem_);
  // evals_.resize(nelem_);
  // assembler_ = std::make_shared<Assembler<double,DIM,-1>>(
  //     mesh_->T_, mesh_->free_map_);
}
template class mfem::MixedStretchGpu<3>; // 3D
template class mfem::MixedStretchGpu<2>; // 2D
